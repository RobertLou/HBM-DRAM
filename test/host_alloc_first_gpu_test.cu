#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../time/timecalculate.h"

__global__ void add(int *a, int *b, int* c, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        c[i] = a[i] + b[i];
    }
}


int main(){
    int *a, *b, *c;
    int *dev_c;
    CTimeCalculate iTimeCal;

    int size = 1000000;

    hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&b, size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&c, size * sizeof(int), hipHostMallocDefault);
    hipMalloc((void **)&dev_c, size * sizeof(int));

    for(int i = 0; i < size; i++){
        a[i] = 1;
        b[i] = 2;
    }

    

    hipStream_t myStream;
    hipStreamCreate(&myStream);


    iTimeCal.StartWork("Kernel Function");
    add<<<(size + 127) / 128, 128, 0, myStream>>>(a, b, dev_c, size);
    hipMemcpyAsync(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost, myStream);
    hipStreamSynchronize(myStream);
    for(int i = 0; i < size; i++){
        if(c[i] != 3){
            std::cout << c[i] << std::endl;
            std::cout << i << std::endl;
        }
    }
    iTimeCal.EndWork("Kernel Function");

    


    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipFree(dev_c);
    hipStreamDestroy(myStream);
    return 0;
}