#include "hip/hip_runtime.h"
#include "embedding_map.h"

__global__ void UpdateOneEmbedding(Parameters *Batch){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for(int j = 0;j < EMBEDDING_DIM;j++){
        Batch[i].a[j] += g * g;
        Batch[i].v[j] -= (c * g * 1.0) / sqrt(Batch[i].a[j]);
    }
}

Parameters* CEmbeddingMap::Get(int Key) {
    std::lock_guard<std::mutex> guard(a_mutex);
    return a_map.at(Key);
};

void CEmbeddingMap::Set(int Key, Parameters* Value) {
    std::lock_guard<std::mutex> guard(a_mutex);
    a_map.insert(std::make_pair(Key, Value)); 
};

void CEmbeddingMap::Erase(int Key)
{
    std::lock_guard<std::mutex> guard(a_mutex);
    a_map.erase(Key);
}

void CEmbeddingMap::InitEmbedding(std::string strFileloc,std::vector<Parameters> &line,int bFirstLineDelete){
    std::ifstream ifDataSet;
    ifDataSet.open(strFileloc);

    std::string strLine;
    char cComma;
    int nKeyTmp;
    std::vector<int> vKey;
    
    if(bFirstLineDelete){
        std::getline(ifDataSet, strLine);
    }
    float a_f,v_f;
    while (std::getline(ifDataSet, strLine))
    {
        std::stringstream ss(strLine);
        Parameters tmp;
        ss >> nKeyTmp;
        ss >> cComma;
        ss >> a_f;
        ss >> cComma;
        ss >> v_f;
        for(int i = 0;i < EMBEDDING_DIM;++i){
            tmp.a[i] = a_f;
            tmp.v[i] = v_f;
        }
        line.emplace_back(tmp);
        vKey.emplace_back(nKeyTmp);
    }
    
    auto iter2 = line.begin();
    for (auto iter1 = vKey.begin(); iter1 != vKey.end(); iter1++) {
        Set(*iter1,&(*iter2));
        iter2++;
    }

    ifDataSet.close();
}

void CEmbeddingMap::UpdateBatch(const std::vector<int>& line, int nCursor, Parameters *Batch, Parameters *BatchAddressGPU, int nCurrentBatchSize, TimeInterval &ti){
    Parameters* tmp;  
    int nBatchCursor = 0;

    //memcpy,将查询到的数据复制到连续的Batch空间中
    clock_gettime(CLOCK_MONOTONIC, &ti.tMemStart);
    for (auto iter = line.cbegin() + nCursor; iter != line.cbegin() + nCursor + nCurrentBatchSize; iter++) {
        tmp = Get(*iter);
        for(int i = 0;i < EMBEDDING_DIM;++i){
            Batch[nBatchCursor].a[i] = tmp->a[i];
            Batch[nBatchCursor].v[i] = tmp->v[i];
        }
        nBatchCursor++;
    }
    clock_gettime(CLOCK_MONOTONIC, &ti.tMemEnd);
    ti.fMemcpyTime1 += ((double)(ti.tMemEnd.tv_sec - ti.tMemStart.tv_sec)*1000000000 + ti.tMemEnd.tv_nsec - ti.tMemStart.tv_nsec)/1000000;


    //计算更新embedding
    hipMemcpy(BatchAddressGPU, Batch, nCurrentBatchSize * sizeof(Parameters), hipMemcpyHostToDevice);
    UpdateOneEmbedding<<<BATCH_SIZE/nDimBlock,nDimBlock>>>(BatchAddressGPU);
    hipMemcpy(Batch, BatchAddressGPU, nCurrentBatchSize * sizeof(Parameters), hipMemcpyDeviceToHost);
        
    //memcpy，将更新后的数据拷回
    nBatchCursor = 0;
    clock_gettime(CLOCK_MONOTONIC, &ti.tMemStart);
    for(auto iter = line.cbegin() + nCursor; iter != line.cbegin() + nCursor + nCurrentBatchSize; iter++) {
        tmp = Get(*iter);
        for(int i = 0;i < EMBEDDING_DIM;++i){
            tmp->a[i] = Batch[nBatchCursor].a[i] ;
            tmp->v[i] = Batch[nBatchCursor].v[i] ;
        }
        nBatchCursor++;
    }
    clock_gettime(CLOCK_MONOTONIC, &ti.tMemEnd);
    ti.fMemcpyTime2 += ((double)(ti.tMemEnd.tv_sec - ti.tMemStart.tv_sec)*1000000000 + ti.tMemEnd.tv_nsec - ti.	tMemStart.tv_nsec)/1000000;
}

void CEmbeddingMap::UpdateWork(const std::vector<int>& line, int start, int end, int workerId)
	{	
		int cursor = start;
		Parameters *Batch= new Parameters[BATCH_SIZE];

		Parameters *BatchAddressGPU;
		TimeInterval ti;

		hipMalloc((void **)&BatchAddressGPU, BATCH_SIZE * sizeof(Parameters));
		while(end - cursor >= BATCH_SIZE){
			UpdateBatch(line, cursor, Batch, BatchAddressGPU, BATCH_SIZE, ti);
			cursor += BATCH_SIZE;
		}
		UpdateBatch(line, cursor, Batch, BatchAddressGPU, end - cursor, ti);
		delete []Batch;
		hipFree(BatchAddressGPU);

		std::cout << "线程" << workerId << "已经结束" << std::endl;
		std::cout << "memcpy time 1:" << ti.fMemcpyTime1 << "ms" << std::endl;		//CPU memcpy time
		std::cout << "memcpy time 2:" << ti.fMemcpyTime2 << "ms" << std::endl;
	}

void CEmbeddingMap::MultiThreadUpdateEV(const std::vector<int>& line) {
    int scope = line.size() / THREAD_NUM;

    std::thread th_arr[THREAD_NUM];

    for (unsigned int i = 0; i < THREAD_NUM - 1; ++i) {
        th_arr[i] = std::thread(&CEmbeddingMap::UpdateWork, this, std::ref(line), i * scope, (i + 1) * scope, i);
    }
    th_arr[THREAD_NUM - 1] = std::thread(&CEmbeddingMap::UpdateWork, this, std::ref(line), (THREAD_NUM - 1) * scope, line.size(), THREAD_NUM - 1);
    for (unsigned int i = 0; i < THREAD_NUM; ++i) {
        th_arr[i].join();
    }
}