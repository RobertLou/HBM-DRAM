#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../time/timecalculate.h"

__global__ void add(int *a, int *b, int* c, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        c[i] = a[i] + b[i];
    }
}


int main(){
    int *a, *b, *c;
    CTimeCalculate iTimeCal;

    int size = 1000000;

    hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&b, size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&c, size * sizeof(int), hipHostMallocDefault);
    

    for(int i = 0; i < size; i++){
        a[i] = 1;
        b[i] = 2;
    }

    hipStream_t myStream;
    hipStreamCreate(&myStream);


    iTimeCal.StartWork("Kernel Function");
    add<<<(size + 127) / 128, 128, 0, myStream>>>(a, b, c, size);

    iTimeCal.EndWork("Kernel Function");

    hipStreamSynchronize(myStream);
    for(int i = 0; i < size; i++){
        if(c[i] != 3){
            std::cout << c[i] << std::endl;
            std::cout << i << std::endl;
        }
    }

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipStreamDestroy(myStream);
    return 0;
}