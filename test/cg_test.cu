#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime.h"
#include ""

#define WARP_SIZE 32

namespace cg = cooperative_groups;

__global__ void add(int *a, int *b, int N){
    cg::thread_block_tile<WARP_SIZE> warp_tile =
        cg::tiled_partition<WARP_SIZE>(cg::this_thread_block());
    const size_t lane_idx = warp_tile.thread_rank();
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        a[i] = i;
        b[i] = warp_tile.meta_group_rank();
    }
}


int main(){

    int *a, *b;
    int *dev_a, *dev_b;


    int size = 1000;

    a = (int *)malloc(sizeof(int) * size);
    b = (int *)malloc(sizeof(int) * size);
    
    hipMalloc((void**)&dev_a, sizeof(int) * size);
    hipMalloc((void**)&dev_b, sizeof(int) * size);

    for(int i = 0; i < size; i++){
        a[i] = 1;
        b[i] = 2;
    }

    hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice);


    add<<<(size + 127) / 128, 128>>>(dev_a, dev_b, size);
    hipDeviceSynchronize();



    hipMemcpy(a, dev_a, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipMemcpy(b, dev_b, sizeof(int) * size, hipMemcpyDeviceToHost);

    for(int i = 0; i < size; i++){
        std::cout << a[i] << "," << b[i] << std::endl;
    }
    return 0;
}