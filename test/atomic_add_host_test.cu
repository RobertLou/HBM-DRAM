#include <iostream>
#include "hip/hip_runtime.h"


__global__ void add1(int *global_counter, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        atomicAdd(global_counter, 1);
    }
}

__global__ void add2(int *global_counter, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        atomicAdd(global_counter, 1);
    }
}

int main(){
    int *h_global_counter, *d_global_counter;
    int *host_counter;
    int size = 10000;

    h_global_counter = (int *)malloc(sizeof(int));
    hipMalloc((void**)&d_global_counter, sizeof(int));
    hipHostAlloc((void **)&host_counter, sizeof(int), hipHostMallocDefault);
    
    *h_global_counter = 0;
    *host_counter = 0;
    hipMemcpy(d_global_counter, h_global_counter, sizeof(int), hipMemcpyHostToDevice);

    for(int i = 0;i < 1000;i++){
        add1<<<(size + 127) / 128, 128>>>(d_global_counter, size);
    }

    for(int i = 0;i < 1000;i++){
        add2<<<(size + 127) / 128, 128>>>(host_counter, size);
    }

    hipDeviceSynchronize();
    hipMemcpy(h_global_counter, d_global_counter, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << *h_global_counter << std::endl;
    std::cout << *host_counter << std::endl;
    return 0;
}