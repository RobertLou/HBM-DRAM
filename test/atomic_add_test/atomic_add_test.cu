#include <iostream>
#include "hip/hip_runtime.h"


__global__ void write(int *count, int *a, int length){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ int lock;
    if(i == 0)
        lock = 0;
    if(i < length){
        bool blocked = true;
        while(blocked) {
            if(0 == atomicCAS(&lock, 0, 1)) {
                atomicAdd(count, 1);
                a[i] = *count;
                atomicExch(&lock, 0);
                blocked = false;
            }
        }
    }
}

int main(){
    int *a, *count, *b;
    int *dev_a, *dev_count;


    a = (int *)malloc(sizeof(int) * 100);
    b = (int *)malloc(sizeof(int) * 100);
    count = (int *)malloc(sizeof(int) * 1);

    
    hipMalloc((void**)&dev_a, sizeof(int) * 100);
    hipMalloc((void**)&dev_count, sizeof(int) * 1);

    *count = 0;
    for(int i = 0; i < 100; i++){
        a[i] = 0;
    }
    hipMemcpy(dev_a, a, sizeof(int) * 100, hipMemcpyHostToDevice);
    hipMemcpy(dev_count, count, sizeof(int) * 1, hipMemcpyHostToDevice);

    write<<<(100 + 31) / 32, 32>>>(dev_count, dev_a, 100);

    hipMemcpy(b, dev_a, sizeof(int) * 100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++){
        std::cout << b[i] << std::endl;
    }

    return 0;
}