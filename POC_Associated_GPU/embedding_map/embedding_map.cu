#include "hip/hip_runtime.h"
#include "embedding_map.h"

Parameters* CEmbeddingMap::Get(int Key){
    std::shared_lock<std::shared_mutex> lock(a_mutex);
	return a_map.at(Key);
};

void CEmbeddingMap::Set(int Key, Parameters* Value){
    std::unique_lock<std::shared_mutex> lock(a_mutex);
	a_map.insert(std::make_pair(Key, Value)); 
};

void CEmbeddingMap::Erase(int key){
	std::unique_lock<std::shared_mutex> lock(a_mutex);
	a_map.erase(key);
}

__global__ void InitEmptyCache(Parameters *GPUEmbeddingAddress){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    GPUEmbeddingAddress[i].key = -1;
}

__global__ void DeviceInitEmbedding(int *locks, Parameters *GPUEmbeddingAddress, Parameters *AllGPUEmbeddings, int length){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < length){
        int key = AllGPUEmbeddings[i].key;
        int cache_id = key % CACHE_NUM;
        int possible_place = cache_id * WAYS;
        bool blocked = true;
        while(blocked) {
            if(0 == atomicCAS(&locks[cache_id], 0, 1)) {
                for(int j = 0;j < WAYS;j++){
                    if(GPUEmbeddingAddress[possible_place + j].key == -1){
                        GPUEmbeddingAddress[possible_place + j].key = key;
                        for(int k = 0; k < EMBEDDING_DIM; k++){
                            GPUEmbeddingAddress[possible_place + j].a[k] = AllGPUEmbeddings[i].a[k];
                            GPUEmbeddingAddress[possible_place + j].v[k] = AllGPUEmbeddings[i].v[k];
                        }
                        GPUEmbeddingAddress[possible_place + j].frequency = 0;
                        break;
                    }
                }
                atomicExch(&locks[cache_id], 0);
                blocked = false;
            }
        }
    }
}

/* __global__ void GatherEmbedding(int *keyBatch, Parameters *GPUEmbeddingAddress, Parameters *deviceGatherResult, int *missCount, int *missIndexList, int *missKeyList, int *lock, int limit){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    if (i == 0){
        *missCount = 0;
        *lock = 0;
    }
    if(i < limit){
        int key = keyBatch[i];
        int cache_id = key % CACHE_NUM;
        int possible_place = cache_id * WAYS;
        for(j = 0; j < WAYS; j++){
            if(GPUEmbeddingAddress[possible_place + j].key == key){
                deviceGatherResult[i].key = key;
                for(int k = 0; k < EMBEDDING_DIM; k++){
                    deviceGatherResult[i].a[k] = GPUEmbeddingAddress[possible_place + j].a[k];
                    deviceGatherResult[i].v[k] = GPUEmbeddingAddress[possible_place + j].v[k];
                }
                atomicAdd(&GPUEmbeddingAddress[possible_place + j].frequency, 1);
                break;
            }
            if(GPUEmbeddingAddress[possible_place + j].key == -1){
                bool blocked = true;
                while(blocked) {
                    if(0 == atomicCAS(lock, 0, 1)) {
                        __threadfence();
                        missKeyList[*missCount] = key;
                        missIndexList[*missCount] = i;
                        atomicAdd(missCount, 1);
                        __threadfence();
                        atomicExch(lock, 0);
                        blocked = false;
                    }
                }
                break;
            }
        }
        if(j == WAYS){
            bool blocked = true;
            while(blocked) {
                if(0 == atomicCAS(lock, 0, 1)) {
                    __threadfence();
                    missKeyList[*missCount] = key;
                    missIndexList[*missCount] = i;
                    atomicAdd(missCount, 1);
                    __threadfence();
                    atomicExch(lock, 0);
                    blocked = false;
                }
            }
        }
    }
} */

__global__ void GatherEmbedding(int *keyBatch, Parameters *GPUEmbeddingAddress, Parameters *deviceGatherResult, int *missCount, int *missIndexList, int *missKeyList, int *lock, int limit){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    if (i == 0){
        *missCount = 0;
        *lock = 0;
    }
    if(i < limit * EMBEDDING_DIM){
        int key_index = i / EMBEDDING_DIM;
        int embedding_index = i % EMBEDDING_DIM;
        int key = keyBatch[key_index];
        int cache_id = key % CACHE_NUM;
        int possible_place = cache_id * WAYS;

        for(j = 0; j < WAYS; j++){
            if(GPUEmbeddingAddress[possible_place + j].key == key){
                if(embedding_index == 0){
                    deviceGatherResult[key_index].key = key;
                    atomicAdd(&GPUEmbeddingAddress[possible_place + j].frequency, 1);
                }
                deviceGatherResult[key_index].a[embedding_index] = GPUEmbeddingAddress[possible_place + j].a[embedding_index];
                deviceGatherResult[key_index].v[embedding_index] = GPUEmbeddingAddress[possible_place + j].v[embedding_index];
                break;
            }
              
            if(embedding_index == 0 && GPUEmbeddingAddress[possible_place + j].key == -1){
                bool blocked = true;
                while(blocked) {
                    if(0 == atomicCAS(lock, 0, 1)) {
                        __threadfence();
                        missKeyList[*missCount] = key;
                        missIndexList[*missCount] = key_index;
                        atomicAdd(missCount, 1);
                        __threadfence();
                        atomicExch(lock, 0);
                        blocked = false;
                    }
                }
                break;
            }
        }
        if(embedding_index == 0  && j == WAYS){
            bool blocked = true;
            while(blocked) {
                if(0 == atomicCAS(lock, 0, 1)) {
                    __threadfence();
                    missKeyList[*missCount] = key;
                    missIndexList[*missCount] = key_index;
                    atomicAdd(missCount, 1);
                    __threadfence();
                    atomicExch(lock, 0);
                    blocked = false;
                }
            }
        }
    }
}

__global__ void GatherMissingEmbedding(int *locks, int *keyBatch, Parameters *GPUEmbeddingAddress, Parameters *deviceGatherResult, int *missIndexList, int *missKeyList, Parameters *deviceMissingEmbedding, int limit){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < limit){
        int key = missKeyList[i];
        int index = missIndexList[i];
        int cache_id = key % CACHE_NUM;
        int possible_place = cache_id * WAYS;

        //写入Result
        deviceGatherResult[index].key =  key;
        deviceGatherResult[index].frequency = 0;
        for(int k = 0; k < EMBEDDING_DIM; k++){
            deviceGatherResult[index].a[k] = deviceMissingEmbedding[i].a[k];
            deviceGatherResult[index].v[k] = deviceMissingEmbedding[i].v[k];
        }
        
        //更新Cache
        bool blocked = true;
        int minFreq = 99999;
        int minPlace = -1;
        while(blocked) {
            if(0 == atomicCAS(&locks[cache_id], 0, 1)) {
                //寻找可替换位置
                for(int j = 0;j < WAYS;j++){
                    if(GPUEmbeddingAddress[possible_place + j].frequency < minFreq){
                        minFreq = GPUEmbeddingAddress[possible_place + j].frequency;
                        minPlace = j;
                    }
                }

                //替换
                GPUEmbeddingAddress[possible_place + minPlace].key = key;
                GPUEmbeddingAddress[possible_place + minPlace].frequency = 0;
                for(int k = 0; k < EMBEDDING_DIM; k++){
                    GPUEmbeddingAddress[possible_place + minPlace].a[k] = deviceMissingEmbedding[i].a[k];
                    GPUEmbeddingAddress[possible_place + minPlace].v[k] = deviceMissingEmbedding[i].v[k];
                }
                __threadfence();
                atomicExch(&locks[cache_id], 0);
                blocked = false;
            }
        }

    }
}

void CEmbeddingMap::InitEmbedding(std::string strFileloc, int bFirstLineDelete){
    std::ifstream ifDataSet;
    ifDataSet.open(strFileloc);

    std::string strLine;
    char cComma;
    int nKeyTmp;
    std::vector<int> vKey;
    
    if(bFirstLineDelete){
        std::getline(ifDataSet, strLine);
    }
    float a_f,v_f;
    while (std::getline(ifDataSet, strLine))
    {
        std::stringstream ss(strLine);
        Parameters tmp;
        ss >> nKeyTmp;
        ss >> cComma;
        ss >> a_f;
        ss >> cComma;
        ss >> v_f;
        for(int i = 0;i < EMBEDDING_DIM;++i){
            tmp.key = nKeyTmp;
            tmp.a[i] = a_f;
            tmp.v[i] = v_f;
            tmp.frequency = 0;
        }
        EmbeddingOnDRAM.emplace_back(tmp);
        vKey.emplace_back(nKeyTmp);
    }

    totalMissCount = 0;
    totalHitCount = 0;
    totalBatch = 0;
    missingBatch = 0;

    //初始化CPU上的embedding map
    auto iter2 = EmbeddingOnDRAM.begin();
    for (auto iter1 = vKey.begin(); iter1 != vKey.end(); iter1++) {
        Set(*iter1,&(*iter2));
        iter2++;
    }

    //初始化组相联Cache的key为-1
    hipMalloc((void **)&GPUEmbeddingAddress, CACHE_SIZE * sizeof(Parameters));
    InitEmptyCache<<<CACHE_SIZE / nDimBlock, nDimBlock>>>(GPUEmbeddingAddress);
    
    hipMalloc((void**)&locks, CACHE_NUM * sizeof(int));
    hipMemset(locks, 0, CACHE_NUM * sizeof(int));
    int length = EmbeddingOnDRAM.size();

    Parameters *AllGPUEmbeddings;
    hipMalloc((void **)&AllGPUEmbeddings, length * sizeof(Parameters));
    hipMemcpy(AllGPUEmbeddings, &EmbeddingOnDRAM[0], length * sizeof(Parameters), hipMemcpyHostToDevice);

    DeviceInitEmbedding<<<length/nDimBlock + 1, nDimBlock>>>(locks, GPUEmbeddingAddress, AllGPUEmbeddings, length);

    ifDataSet.close();
}


void CEmbeddingMap::GatherBatch(const std::vector<int>& line, int cursor, Parameters *gatherResult, int currentBatchSize){ 
    //将Batch中的key拷贝到GPU
    int *keyBatch;
    hipMalloc((void **)&keyBatch, currentBatchSize * sizeof(int));
    hipMemcpy(keyBatch, &line[cursor], currentBatchSize * sizeof(int), hipMemcpyHostToDevice);

    clock_gettime(CLOCK_MONOTONIC, &tStart);

    //创建查找到的embedding数据存储的空间
    Parameters *deviceGatherResult;
    hipMalloc((void **)&deviceGatherResult, currentBatchSize * sizeof(Parameters));

    //创建MissList的空间
    int *devMissLock;
    hipMalloc((void **)&devMissLock, sizeof(int));

    int *deviceMissKeyList, *missKeyList, *deviceMissIndexList;
    hipMalloc((void **)&deviceMissKeyList, currentBatchSize * sizeof(int));
    hipMalloc((void **)&deviceMissIndexList, currentBatchSize * sizeof(int));
    missKeyList = new int[currentBatchSize]();

    int missCount = 0, *devMissCount;
    hipMalloc((void **)&devMissCount, sizeof(int));
    //Gather 
    GatherEmbedding<<<(BATCH_SIZE * EMBEDDING_DIM + nDimBlock - 1) / nDimBlock, nDimBlock>>>(keyBatch, GPUEmbeddingAddress, deviceGatherResult, devMissCount, deviceMissIndexList, deviceMissKeyList, devMissLock, currentBatchSize);
    
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC, &tEnd);
    hitTime += ((double)(tEnd.tv_sec - tStart.tv_sec)*1000000000 + tEnd.tv_nsec - tStart.tv_nsec)/1000000;

    //如果有缺少的，从CPU上拉取
    clock_gettime(CLOCK_MONOTONIC, &tStart);
    hipMemcpy(&missCount, devMissCount, sizeof(int), hipMemcpyDeviceToHost);
    totalBatch++;  
    clock_gettime(CLOCK_MONOTONIC, &tEnd);
    statusMemcpyTime += ((double)(tEnd.tv_sec - tStart.tv_sec)*1000000000 + tEnd.tv_nsec - tStart.tv_nsec)/1000000;

    if(missCount > 0){        
        clock_gettime(CLOCK_MONOTONIC, &tStart);
        missingBatch++;
        Parameters *missingEmbedding, *deviceMissingEmbedding;
        hipMalloc(&deviceMissingEmbedding, missCount * sizeof(Parameters));
        missingEmbedding = new Parameters[missCount];
        
        
        hipMemcpy(missKeyList, deviceMissKeyList, sizeof(int) * missCount, hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC, &tEnd);
        memcpyTime += ((double)(tEnd.tv_sec - tStart.tv_sec)*1000000000 + tEnd.tv_nsec - tStart.tv_nsec)/1000000;

        clock_gettime(CLOCK_MONOTONIC, &tStart);
        //从CPU中查找缺失的Embedding
        //TODO::修改为多线程查找
        for(int i = 0; i < missCount; i++){
            Parameters *tmp;
            tmp = Get(missKeyList[i]);
            missingEmbedding[i].key = tmp->key;
            for(int j = 0;j < EMBEDDING_DIM; j++){
                missingEmbedding[i].a[j] = tmp->a[j];
                missingEmbedding[i].v[j] = tmp->v[j];
            }
            missingEmbedding[i].frequency = tmp->frequency;
        }
        clock_gettime(CLOCK_MONOTONIC, &tEnd);
        lookUpTime += ((double)(tEnd.tv_sec - tStart.tv_sec)*1000000000 + tEnd.tv_nsec - tStart.tv_nsec)/1000000;
        
        //将查询结果拷上GPU
        clock_gettime(CLOCK_MONOTONIC, &tStart);
        hipMemcpy(deviceMissingEmbedding, missingEmbedding, missCount * sizeof(Parameters), hipMemcpyHostToDevice);
        GatherMissingEmbedding<<<(missCount + nDimBlock - 1) / nDimBlock, nDimBlock>>>(locks, keyBatch, GPUEmbeddingAddress, deviceGatherResult, deviceMissIndexList, deviceMissKeyList, deviceMissingEmbedding, missCount);
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tEnd);
        memcpyTime += ((double)(tEnd.tv_sec - tStart.tv_sec)*1000000000 + tEnd.tv_nsec - tStart.tv_nsec)/1000000;

        delete []missingEmbedding;
        hipFree(deviceMissingEmbedding);
    }

    //将结果拷贝回CPU检验
    hipMemcpy(&gatherResult[cursor], deviceGatherResult, currentBatchSize * sizeof(Parameters), hipMemcpyDeviceToHost);


    totalHitCount += currentBatchSize - missCount;
    totalMissCount += missCount;


    delete []missKeyList;
    hipFree(devMissLock);
    hipFree(deviceMissKeyList);
    hipFree(deviceMissIndexList);
    hipFree(devMissCount);
    hipFree(deviceGatherResult);
    hipFree(keyBatch);
}

void CEmbeddingMap::GatherWork(const std::vector<int>& line, Parameters *gatherResult){
    int cursor = 0;
    int end = line.size();
    hitTime = 0;
    statusMemcpyTime = 0;
    lookUpTime = 0;
    memcpyTime = 0;

    while(end - cursor >= BATCH_SIZE){
        GatherBatch(line, cursor, gatherResult, BATCH_SIZE);
        cursor += BATCH_SIZE;
    }
    GatherBatch(line, cursor, gatherResult, end - cursor);
}

float CEmbeddingMap::GetHitRate(){
    return totalHitCount / (totalHitCount + totalMissCount);
}

float CEmbeddingMap::GetMissingBatchRate(){
    return missingBatch / totalBatch;
}

float CEmbeddingMap::GetHitTime(){
    return hitTime;
}

float CEmbeddingMap::GetStatusMemcpyTime(){
    return statusMemcpyTime;
}

float CEmbeddingMap::GetLookUpTime(){
    return lookUpTime;
}

float CEmbeddingMap::GetMemcpyTime(){
    return memcpyTime;
}

void CEmbeddingMap::MoveAllEmbeddings(Parameters *CPUEmbeddingAddress){
    hipMemcpy(CPUEmbeddingAddress, GPUEmbeddingAddress, CACHE_SIZE * sizeof(Parameters), hipMemcpyDeviceToHost);
}

void CEmbeddingMap::DeleteEmbedding(){
    hipFree(locks);
    hipFree(GPUEmbeddingAddress);
}

