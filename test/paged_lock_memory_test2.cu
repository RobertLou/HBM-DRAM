#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../time/timecalculate.h"

#define SIZE (1000 * 1024 * 1024)
#define N 1

const int block_dim = 128;

__global__ void Init(int *a, int length){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < length){
        a[i] = i;
    }
}

void cuda_malloc_test(int size, bool up){
    CTimeCalculate iTimeCal;
    int *a, *dev_a;

    a = (int *)malloc(size * sizeof(int));
    hipMalloc((void **)&dev_a, size * sizeof(int));
    for(int i = 0; i < size;i++){
        a[i] = 0;
    }

    iTimeCal.StartWork("Cuda malloc");
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    Init<<<(size + block_dim - 1) / block_dim, block_dim>>>(dev_a, size);
    hipDeviceSynchronize();
    iTimeCal.EndWork("Cuda malloc");

    free(a);
    hipFree(dev_a);
}

void host_malloc_test(int size, bool up){
    CTimeCalculate iTimeCal;
    int *a, *dev_a;

    hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocDefault);
    hipMalloc((void **)&dev_a, size * sizeof(int));
    for(int i = 0; i < size;i++){
        a[i] = 0;
    }
    
    iTimeCal.StartWork("host malloc");
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    Init<<<(size + block_dim - 1) / block_dim, block_dim>>>(dev_a, size);
    hipDeviceSynchronize();
    iTimeCal.EndWork("host malloc");

    hipHostFree(a);
    hipFree(dev_a);
}

void host_malloc_test2(int size, bool up){
    CTimeCalculate iTimeCal;
    int *a;

    hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocDefault);
    for(int i = 0; i < size;i++){
        a[i] = 0;
    }

    iTimeCal.StartWork("host malloc");
    Init<<<(size + block_dim - 1) / block_dim, block_dim>>>(a, size);
    hipDeviceSynchronize();
    std::cout << a[3] << std::endl;
    iTimeCal.EndWork("host malloc");

    hipHostFree(a);
}

int main(){
    int *b;
    hipMalloc((void **)&b, sizeof(int));
    hipFree(b);
    hipHostAlloc((void **)&b, sizeof(int), hipHostMallocDefault);
    hipHostFree(b);

    //cuda_malloc_test(SIZE, true);
    //host_malloc_test(SIZE, true);
    host_malloc_test2(SIZE, true);

    return 0;
}