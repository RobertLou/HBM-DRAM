#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../time/timecalculate.h"

#define SIZE (8 * 1024 * 1024)
#define N 1

const int block_dim = 128;

__global__ void Init(int *a, int length){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < length){
        a[i] = i;
    }
}

void cuda_malloc_test(int size, bool up){
    CTimeCalculate iTimeCal;
    int *a, *dev_a;

    a = (int *)malloc(size * sizeof(int));
    hipMalloc((void **)&dev_a, size * sizeof(int));
    for(int i = 0; i < size;i++){
        a[i] = 0;
    }

    iTimeCal.StartWork("Cuda malloc");
    for(int i = 0; i < N; i++){
        hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    }
    Init<<<(size + block_dim - 1) / block_dim, block_dim>>>(dev_a, size);
    hipDeviceSynchronize();
    iTimeCal.EndWork("Cuda malloc");

    free(a);
    hipFree(dev_a);
}

void host_malloc_test(int size, bool up){
    CTimeCalculate iTimeCal;
    int *a, *dev_a;

    //hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocDefault);
    //hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocPortable);
    //hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocMapped);
    hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocWriteCombined);
    hipMalloc((void **)&dev_a, size * sizeof(int));
    for(int i = 0; i < size;i++){
        a[i] = 0;
    }
    
    iTimeCal.StartWork("host malloc");
    for(int i = 0; i < N; i++){
        hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    }
    Init<<<(size + block_dim - 1) / block_dim, block_dim>>>(dev_a, size);
    hipDeviceSynchronize();
    iTimeCal.EndWork("host malloc");

    hipHostFree(a);
    hipFree(dev_a);
}

void host_malloc_test2(int size, bool up){
    CTimeCalculate iTimeCal;
    int *a;

    //hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocDefault);
    //hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocPortable);
    //hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocMapped);
    hipHostAlloc((void **)&a, size * sizeof(int), hipHostMallocWriteCombined);
    for(int i = 0; i < size;i++){
        a[i] = 0;
    }
    
    iTimeCal.StartWork("host malloc");
    Init<<<(size + block_dim - 1) / block_dim, block_dim>>>(a, size);
    hipDeviceSynchronize();
    iTimeCal.EndWork("host malloc");

    std::cout << a[32] << std::endl;

    hipHostFree(a);
}


int main(){
    int *b;
    hipMalloc((void **)&b, sizeof(int));
    hipFree(b);
    hipHostAlloc((void **)&b, sizeof(int), hipHostMallocDefault);
    hipHostFree(b);

    //cuda_malloc_test(SIZE, true);
    host_malloc_test(SIZE, true);
    //host_malloc_test2(SIZE, true);

    return 0;
}