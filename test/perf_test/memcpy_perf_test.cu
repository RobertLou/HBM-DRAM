#include "hip/hip_runtime.h"
/*
Target: Copy Device data to Host
        Compare direct copy or using memcpy
*/
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../../time/timecalculate.h"

__global__ void copy(int *dst, int *src, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        dst[i] = src[i];
    }
}


int main(){
    const int N = 100000000;
    int *h_src, *d_src;
    int *h_dst1, *d_dst1;
    int *h_dst2;

    hipMalloc((void **)&d_src, N * sizeof(int));
    hipMalloc((void **)&d_dst1, N * sizeof(int));
    
    hipHostAlloc((void **)&h_src, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&h_dst1, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&h_dst2, N * sizeof(int), hipHostMallocDefault);
    for (int i = 0; i < N;++i){
        h_src[i] = i;
    }
    hipMemcpy(d_src, h_src, N * sizeof(int), hipMemcpyHostToDevice);
    CTimeCalculate iTimeCal;

    iTimeCal.StartWork("Copy to Device then copy to Host");
    copy<<<(N + 127) / 128, 128>>>(d_dst1, d_src, N);
    hipMemcpyAsync(h_dst1, d_dst1, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    iTimeCal.EndWork("Copy to Device then copy to Host");
    
    iTimeCal.StartWork("Direct Copy to Host");
    copy<<<(N + 127) / 128, 128>>>(h_dst2, d_src, N);
    hipDeviceSynchronize();
    iTimeCal.EndWork("Direct Copy to Host");

    std::cout << h_dst1[13] << std::endl;
    std::cout << h_dst2[13] << std::endl;

    hipFree(d_src);
    hipFree(d_dst1);
    
    hipHostFree(h_src);
    hipHostFree(h_dst1);
    hipHostFree(h_dst2);
    
    return 0;
}