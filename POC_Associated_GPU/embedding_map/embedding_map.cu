#include "hip/hip_runtime.h"
#include "embedding_map.h"

__global__ void InitEmptyCache(Parameters *GPUEmbeddingAddress){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    GPUEmbeddingAddress[i].key = -1;
}

__global__ void HostInitEmbedding(Parameters *GPUEmbeddingAddress, Parameters *AllGPUEmbeddings, int length){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < length){
        int key = AllGPUEmbeddings[i].key;
        int cache_id = key / CACHE_NUM * WAYS;
        for(int j = 0;j < WAYS;j++){
            if(GPUEmbeddingAddress[cache_id + j].key == -1){
                GPUEmbeddingAddress[cache_id + j].key = i;
                for(int k = 0; k < EMBEDDING_DIM; k++){
                    GPUEmbeddingAddress[cache_id + j].a[k] = AllGPUEmbeddings[i].a[k];
                    GPUEmbeddingAddress[cache_id + j].v[k] = AllGPUEmbeddings[i].v[k];
                }
                break;
            }
        }
    }
}

__global__ void GatherEmbedding(int *keyBatch, Parameters *GPUEmbeddingAddress, Parameters *devicegatherResult, int currentBatchSize){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    if(i < currentBatchSize){
        int key = keyBatch[i];
        int cache_id = key / CACHE_NUM * WAYS;
        
        for(j = 0; j < WAYS; j++){
            if(GPUEmbeddingAddress[cache_id + j].key == key){
                devicegatherResult[i].key = key;
                for(int k = 0; k < EMBEDDING_DIM; k++){
                    devicegatherResult[i].a[k] = GPUEmbeddingAddress[cache_id + j].a[k];
                    devicegatherResult[i].v[k] = GPUEmbeddingAddress[cache_id + j].v[k];
                }
                break;
            }
        }
        if(j == WAYS){
            for(int k = 0; k < EMBEDDING_DIM; k++){
                devicegatherResult[i].a[k] = -1;
                devicegatherResult[i].v[k] = -1;
            }
        }
    }
}

void CEmbeddingMap::InitEmbedding(std::string strFileloc,std::vector<Parameters> &line,int bFirstLineDelete){
    std::ifstream ifDataSet;
    ifDataSet.open(strFileloc);

    std::string strLine;
    char cComma;
    int nKeyTmp;
    std::vector<int> vKey;
    
    if(bFirstLineDelete){
        std::getline(ifDataSet, strLine);
    }
    float a_f,v_f;
    while (std::getline(ifDataSet, strLine))
    {
        std::stringstream ss(strLine);
        Parameters tmp;
        ss >> nKeyTmp;
        ss >> cComma;
        ss >> a_f;
        ss >> cComma;
        ss >> v_f;
        for(int i = 0;i < EMBEDDING_DIM;++i){
            tmp.key = nKeyTmp;
            tmp.a[i] = a_f;
            tmp.v[i] = v_f;
            tmp.frequency = 0;
        }
        line.emplace_back(tmp);
        vKey.emplace_back(nKeyTmp);
    }
    //初始化组相联Cache的key为-1
    hipMalloc((void **)&GPUEmbeddingAddress, CACHE_SIZE * sizeof(Parameters));
    InitEmptyCache<<<CACHE_SIZE / nDimBlock, nDimBlock>>>(GPUEmbeddingAddress);

    int length = line.size();

    Parameters *AllGPUEmbeddings;
    hipMalloc((void **)&AllGPUEmbeddings, length * sizeof(Parameters));
    hipMemcpy(AllGPUEmbeddings, &line[0], length * sizeof(Parameters), hipMemcpyHostToDevice);

    HostInitEmbedding<<<length/nDimBlock + 1, nDimBlock>>>(GPUEmbeddingAddress, AllGPUEmbeddings, length);

    ifDataSet.close();
}


void CEmbeddingMap::GatherBatch(const std::vector<int>& line, int cursor, Parameters *gatherResult, int currentBatchSize){ 


    //将Batch中的key拷贝到GPU
    int *keyBatch;
    hipMalloc((void **)&keyBatch, currentBatchSize * sizeof(int));
    hipMemcpy(keyBatch, &line[cursor], currentBatchSize * sizeof(int), hipMemcpyHostToDevice);

    //创建查找到的embedding数据存储的空间
    Parameters *devicegatherResult;
    hipMalloc((void **)&devicegatherResult, currentBatchSize * sizeof(Parameters));

    //Gather 
    GatherEmbedding<<<BATCH_SIZE/nDimBlock, nDimBlock>>>(keyBatch, GPUEmbeddingAddress, devicegatherResult, currentBatchSize);
    hipDeviceSynchronize();

    //将结果拷贝回CPU检验
    hipMemcpy(&gatherResult[cursor], devicegatherResult, currentBatchSize * sizeof(Parameters), hipMemcpyDeviceToHost);
    
    hipFree(devicegatherResult);
    hipFree(keyBatch);
}

void CEmbeddingMap::GatherWork(const std::vector<int>& line, Parameters *gatherResult){
    int cursor = 0;
    int end = line.size();

    while(end - cursor >= BATCH_SIZE){
        GatherBatch(line, cursor, gatherResult, BATCH_SIZE);
        cursor += BATCH_SIZE;
    }
    GatherBatch(line, cursor, gatherResult, end - cursor);
}

void CEmbeddingMap::DeleteEmbedding(){
    hipFree(GPUEmbeddingAddress);
}