#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../time/timecalculate.h"

#define MAX_THREADS_PER_BLOCK 128
#define MAX_ELEMENTS_PER_BLOCK (MAX_THREADS_PER_BLOCK * 2)

__global__ void parallel_large_scan_kernel(int *data, int *prefix_sum, int N, int *sums)
{
    __shared__ int tmp[MAX_ELEMENTS_PER_BLOCK];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
    int leaf_num = MAX_ELEMENTS_PER_BLOCK;

    tmp[tid * 2] = tid * 2 + block_offset < N ? data[tid * 2 + block_offset] : 0;
    tmp[tid * 2 + 1] = tid * 2 + 1 + block_offset < N ? data[tid * 2 + 1 + block_offset] : 0;
    __syncthreads();

    int offset = 1;
    for (int d = leaf_num >> 1; d > 0; d >>= 1)
    {
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            tmp[bi] += tmp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (tid == 0)
    {
        sums[bid] = tmp[leaf_num - 1];
        tmp[leaf_num - 1] = 0;
    }
    __syncthreads();

    for (int d = 1; d < leaf_num; d *= 2)
    {
        offset >>= 1;
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            float v = tmp[ai];
            tmp[ai] = tmp[bi];
            tmp[bi] += v;
        }
        __syncthreads();
    }

    if (tid * 2 + block_offset < N)
    {
        prefix_sum[tid * 2 + block_offset] = tmp[tid * 2];
    }
    if (tid * 2 + 1 + block_offset < N)
    {
        prefix_sum[tid * 2 + 1 + block_offset] = tmp[tid * 2 + 1];
    }
}

__global__ void add_kernel(int *prefix_sum, int *value, int N)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
    int ai = tid + block_offset;
    int bi = tid + (MAX_ELEMENTS_PER_BLOCK >> 1) + block_offset;

    if (ai < N)
    {
        prefix_sum[ai] += value[bid];
    }
    if (bi < N)
    {
        prefix_sum[bi] += value[bid];
    }
}

void recursive_scan(int *d_data, int *d_prefix_sum, int N)
{
    int block_num = (N - 1) / MAX_ELEMENTS_PER_BLOCK + 1;
    int *d_sums, *d_sums_prefix_sum;  // 用来保存block数组和、数组和的前缀和
    hipMalloc(&d_sums, block_num * sizeof(int));
    hipMalloc(&d_sums_prefix_sum, block_num * sizeof(int));

    parallel_large_scan_kernel<<<block_num, MAX_THREADS_PER_BLOCK>>>(d_data, d_prefix_sum, N, d_sums);

    if (block_num != 1)
    {
        recursive_scan(d_sums, d_sums_prefix_sum, block_num);
        add_kernel<<<block_num, MAX_THREADS_PER_BLOCK>>>(d_prefix_sum, d_sums_prefix_sum, N);
    }
}


int main(){
    int *in, *out;
    int *d_in, *d_out;
    CTimeCalculate iTimeCal;

    int size = 111111;

    in = (int *)malloc(sizeof(int) * size);
    out = (int *)malloc(sizeof(int) * (size + 1));
    
    hipMalloc((void**)&d_in, sizeof(int) * size);
    hipMalloc((void**)&d_out, sizeof(int) * (size + 1));

    for(int i = 0; i < size; i++){
        in[i] = 1;
    }

    hipMemcpy(d_in, in, sizeof(int) * size, hipMemcpyHostToDevice);


    iTimeCal.StartWork("Kernel Function");
    recursive_scan(d_in, d_out, size + 1);
    hipDeviceSynchronize();
    iTimeCal.EndWork("Kernel Function");


    hipMemcpy(out, d_out, sizeof(int) * (size + 1), hipMemcpyDeviceToHost);

    for(int i = 0; i <= size; i++){
        if(out[i] != i){
            std::cout << i << std::endl;
            std::cout << out[i] << std::endl;
        }
    }

    hipFree(d_in);
    hipFree(d_out);
    delete[] in;
    delete[] out;
    return 0;
}