#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../time/timecalculate.h"

int main(){
    int *a, *b;
    int *dev_a, *dev_b;
    CTimeCalculate iTimeCal;

    a = (int *)malloc(sizeof(int) * 1);
    b = (int *)malloc(sizeof(int) * 100000);
    
    hipMalloc((void**)&dev_a, sizeof(int) * 1);
    hipMalloc((void**)&dev_b, sizeof(int) * 100000);

    iTimeCal.StartWork("memcpy 1 int");
    hipMemcpy(dev_a, a, sizeof(int) * 1, hipMemcpyHostToDevice);
    iTimeCal.EndWork("memcpy 1 int");

    iTimeCal.StartWork("memcpy 100000 int");
    hipMemcpy(dev_b, b, sizeof(int) * 100000, hipMemcpyHostToDevice);
    iTimeCal.EndWork("memcpy 100000 int");
    return 0;
}