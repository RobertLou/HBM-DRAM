#include <iostream>
#include "hip/hip_runtime.h"


__global__ void add(){
    //atomicAdd(global_counter, 1);
}

int main(){
    int h_global_counter, *d_global_counter;
    int size = 12968;

    hipMalloc((void**)&d_global_counter, sizeof(int));
    
    for(int i = 0;i < size;i++){
        add<<<1, 1>>>();
    }

    hipDeviceSynchronize();
    hipMemcpy(&h_global_counter, d_global_counter, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << h_global_counter << std::endl;

    int runtime_version;
    hipRuntimeGetVersion(&runtime_version);
    std::cout << "CUDA Runtime Version: " << runtime_version << std::endl;
    return 0;
}