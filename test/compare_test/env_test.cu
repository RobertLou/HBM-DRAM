#include <iostream>
#include "hip/hip_runtime.h"


__global__ void add(int *global_counter){
    atomicAdd(global_counter, 1);
}

int main(){
    int h_global_counter, *d_global_counter;
    int size = 12968;

    hipMalloc((void**)&d_global_counter, sizeof(int));
    
    for(int i = 0;i < size;i++){
        add<<<1, 1>>>(d_global_counter);
    }

    hipDeviceSynchronize();
    hipMemcpy(&h_global_counter, d_global_counter, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << h_global_counter << std::endl;

    int runtime_version;
    hipRuntimeGetVersion(&runtime_version);
    std::cout << "CUDA Runtime Version: " << runtime_version << std::endl;
    return 0;
}