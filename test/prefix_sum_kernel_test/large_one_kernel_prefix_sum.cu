#include "hip/hip_runtime.h"
#include "../../time/timecalculate.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_cooperative_groups.h>
#include <iostream>

#define MAX_THREADS_PER_BLOCK 256
#define MAX_ELEMENTS_PER_BLOCK (MAX_THREADS_PER_BLOCK * 2)

namespace cg = cooperative_groups;

__global__ void parallel_large_scan_kernel(int *data, int *prefix_sum, int N, int *sums)
{
    __shared__ int tmp[MAX_ELEMENTS_PER_BLOCK];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
    int leaf_num = MAX_ELEMENTS_PER_BLOCK;

    tmp[tid * 2] = tid * 2 + block_offset < N ? data[tid * 2 + block_offset] : 0;
    tmp[tid * 2 + 1] = tid * 2 + 1 + block_offset < N ? data[tid * 2 + 1 + block_offset] : 0;
    __syncthreads();

    int offset = 1;
    for (int d = leaf_num >> 1; d > 0; d >>= 1)
    {
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            tmp[bi] += tmp[ai];
        }
        offset *= 2;
        __syncthreads();
    }

    if (tid == 0)
    {
        sums[bid] = tmp[leaf_num - 1];
        tmp[leaf_num - 1] = 0;
    }
    __syncthreads();

    for (int d = 1; d < leaf_num; d *= 2)
    {
        offset >>= 1;
        if (tid < d)
        {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            int v = tmp[ai];
            tmp[ai] = tmp[bi];
            tmp[bi] += v;
        }
        __syncthreads();
    }

    if (tid * 2 + block_offset < N)
    {
        prefix_sum[tid * 2 + block_offset] = tmp[tid * 2];
    }
    if (tid * 2 + 1 + block_offset < N)
    {
        prefix_sum[tid * 2 + 1 + block_offset] = tmp[tid * 2 + 1];
    }

    __threadfence();
    int block_num = (N - 1) / MAX_ELEMENTS_PER_BLOCK + 1;
    if (bid == 0 && block_num != 1)
    {
        __shared__ int tmp2[MAX_ELEMENTS_PER_BLOCK];

        tmp2[tid * 2] = tid * 2 < block_num ? sums[tid * 2] : 0;
        tmp2[tid * 2 + 1] = tid * 2 + 1 < block_num ? sums[tid * 2 + 1] : 0;
        __syncthreads();

        offset = 1;
        for (int d = leaf_num >> 1; d > 0; d >>= 1)
        {
            if (tid < d)
            {
                int ai = offset * (2 * tid + 1) - 1;
                int bi = offset * (2 * tid + 2) - 1;
                tmp2[bi] += tmp2[ai];
            }
            offset *= 2;
            __syncthreads();
        }

        if (tid == 0)
        {
            tmp2[leaf_num - 1] = 0;
        }
        __syncthreads();

        for (int d = 1; d < leaf_num; d *= 2)
        {
            offset >>= 1;
            if (tid < d)
            {
                int ai = offset * (2 * tid + 1) - 1;
                int bi = offset * (2 * tid + 2) - 1;

                int v = tmp2[ai];
                tmp2[ai] = tmp2[bi];
                tmp2[bi] += v;
            }
            __syncthreads();
        }

        if (tid * 2 < block_num)
        {
            sums[tid * 2] = tmp2[tid * 2];
        }
        if (tid * 2 + 1 < block_num)
        {
            sums[tid * 2 + 1] = tmp2[tid * 2 + 1];
        }
    }

    if (block_num > 1)
    {

        cg::grid_group grid = cg::this_grid();
        grid.sync();
        int ai = tid + block_offset;
        int bi = tid + (MAX_ELEMENTS_PER_BLOCK >> 1) + block_offset;

        if (ai < N)
        {
            prefix_sum[ai] += sums[bid];
        }
        if (bi < N)
        {
            prefix_sum[bi] += sums[bid];
        }
    }
}

void recursive_scan(int *d_data, int *d_prefix_sum, int N)
{
    int block_num = (N - 1) / MAX_ELEMENTS_PER_BLOCK + 1;
    int *d_sums; // 用来保存block数组和、数组和的前缀和

    hipHostAlloc(&d_sums, block_num * sizeof(int), hipHostMallocDefault);

    void *kernelArgs[] = {(void *)&d_data,
                          (void *)&d_prefix_sum,
                          (void *)&N,
                          (void *)&d_sums};
    hipLaunchCooperativeKernel((void *)parallel_large_scan_kernel, block_num, MAX_THREADS_PER_BLOCK, kernelArgs);
    hipHostFree(d_sums);
}

int main()
{
    int *in, *out;
    CTimeCalculate iTimeCal;

    int size = 1 << 16 - 1;

    hipHostAlloc(&in, sizeof(int) * size, hipHostMallocDefault);
    hipHostAlloc(&out, sizeof(int) * (size + 1), hipHostMallocDefault);

    for (int i = 0; i < size; i++)
    {
        in[i] = 1;
    }

    iTimeCal.StartWork("Kernel Function");
    recursive_scan(in, out, size + 1);
    hipDeviceSynchronize();
    iTimeCal.EndWork("Kernel Function");

    for (int i = 0; i <= size; i++)
    {
        if (out[i] != i)
        {
            std::cout << i << std::endl;
            std::cout << out[i] << std::endl;
        }
    }

    hipHostFree(in);
    hipHostFree(out);
    return 0;
}