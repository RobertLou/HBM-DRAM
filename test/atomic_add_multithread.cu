#include <iostream>
#include <thread>
#include "hip/hip_runtime.h"


__global__ void add(int *global_counter, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        atomicAdd(global_counter, 1);
    }
}

void Add(int *global_counter, int N) {
    add<<<(N + 127) / 128, 128>>>(global_counter, N);
}


int main(){

    int *h_global_counter, *d_global_counter;
    int size = 10000;

    h_global_counter = (int *)malloc(sizeof(int));
    hipMalloc((void**)&d_global_counter, sizeof(int));

    *h_global_counter = 0;
    hipMemcpy(d_global_counter, h_global_counter, sizeof(int), hipMemcpyHostToDevice);

    std::thread thread1(Add, d_global_counter, size);
    std::thread thread2(Add, d_global_counter, size);
    std::thread thread3(Add, d_global_counter, size);
    std::thread thread4(Add, d_global_counter, size);

    thread1.join();
    thread2.join();
    thread3.join();
    thread4.join();

    hipDeviceSynchronize();
    hipMemcpy(h_global_counter, d_global_counter, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << *h_global_counter << std::endl;
    return 0;
}